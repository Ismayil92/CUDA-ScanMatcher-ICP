#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "scanmatch.h"
#include "svd3.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) utilityCore::checkCUDAError(msg, __LINE__)

#define DEBUG false

/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

/*! Size of the starting area in simulation space. 
 * FOR SINE TEST: 2.f
 * FOR ELEPHANT OBJ: 
 * FOR BUDDHA OBJ: 1 << 2;
 * FOR WAYMO DATASET: 1 << 5;
*/

#define scene_scale 1 << 3

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

glm::vec3 *dev_pos;
glm::vec3 *dev_rgb;

pointcloud* target_pc;
pointcloud* src_pc;

//OCTREE pointer (all octnodes lie in device memory)
Octree* octree;

/******************
* initSimulation *
******************/
/**
* Initialize memory, update some globals
*/
void ScanMatch::initSimulationCPU(int N, std::vector<glm::vec3> coords) {
  numObjects = N;

  //Setup and initialize source and target pointcloud
  src_pc = new pointcloud(false, numObjects, false);
  src_pc->initCPU();
  target_pc = new pointcloud(true, numObjects, false);
  target_pc->initCPU();
}

void ScanMatch::initSimulationGPU(int N , std::vector<glm::vec3> coords) {
  numObjects = N;

  //Setup and initialize source and target pointcloud
  src_pc = new pointcloud(false, numObjects, true);
  src_pc->initGPU(coords);
  target_pc = new pointcloud(true, numObjects, true);
  target_pc->initGPU(coords);
}

void ScanMatch::initSimulationGPUOCTREE(int N , std::vector<glm::vec3> coords) {
  numObjects = N;
  //First create the octree 
  octree = new Octree(glm::vec3(0.f, 0.f, 0.f), 4.f, coords);
  octree->create();
  octree->compact();

  //Extract Final Data from it
  int numElts = octree->stackPointer;
  int numCoords = octree->compactedCoords.size();
  glm::vec3* octoCoords = octree->compactedCoords.data();
  OctNode* octNodePool = octree->octNodePool.data();


	printf("NUM ELTS %d \n", numCoords);
	printf("numOBJECTS %d \n", numObjects);
	
	//Send stuff to device
	glm::vec3* dev_octoCoords;
	glm::vec3* dev_octNodePool;

	hipMalloc((void**)&dev_octoCoords, numObjects * sizeof(glm::vec3));
	hipMalloc((void**)&dev_octNodePool, numElts * sizeof(OctNode));
	utilityCore::checkCUDAError("hipMalloc octor failed", __LINE__);

	hipMemcpy(dev_octoCoords, octoCoords, numObjects * sizeof(glm::vec3), hipMemcpyHostToDevice);
	hipMemcpy(dev_octNodePool, octNodePool, numElts * sizeof(OctNode), hipMemcpyHostToDevice);
	utilityCore::checkCUDAError("hipMalloc octor failed", __LINE__);
	
  //Setup and initialize source and target pointcloud
  src_pc = new pointcloud(false, numObjects, true);
  src_pc->initGPU(coords);
  target_pc = new pointcloud(true, numObjects, true);
  target_pc->initGPUWOCTREE(dev_octoCoords);
  //target_pc->initGPU(coords);
}

/******************
* copyPointCloudToVBO *
******************/

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void ScanMatch::copyPointCloudToVBO(float *vbodptr_positions, float *vbodptr_rgb, bool usecpu) {

	if (usecpu) { //IF CPU
	  src_pc->pointCloudToVBOCPU(vbodptr_positions, vbodptr_rgb, scene_scale);
	  target_pc->pointCloudToVBOCPU(vbodptr_positions + 4*numObjects, vbodptr_rgb + 4*numObjects, scene_scale);
	}
	else { //IF GPU
		src_pc->pointCloudToVBOGPU(vbodptr_positions, vbodptr_rgb, scene_scale);
		target_pc->pointCloudToVBOGPU(vbodptr_positions + 4*numObjects, vbodptr_rgb + 4*numObjects, scene_scale);
	}
}


/******************
* stepSimulation *
******************/

void ScanMatch::endSimulation() {
	src_pc->~pointcloud();
	target_pc->~pointcloud();
}

/******************
* CPU SCANMATCHING *
******************/

/**
 * Main Algorithm for Running ICP on the CPU
 * Finds homogenous transform between src_pc and target_pc 
*/
void ScanMatch::stepICPCPU() {
	//1: Find Nearest Neigbors and Reshuffle
	float* dist = new float[numObjects];
	int* indicies = new int[numObjects];
#if DEBUG
	printf("NEAREST NEIGHBORS \n");
#endif // DEBUG

	ScanMatch::findNNCPU(src_pc, target_pc, dist, indicies, numObjects);
#if DEBUG
	printf("RESHUFFLE\n");
#endif // DEBUG

	ScanMatch::reshuffleCPU(target_pc, indicies, numObjects);

	//2: Find Best Fit Transformation
	glm::mat3 R;
	glm::vec3 t;
	ScanMatch::bestFitTransform(src_pc, target_pc, numObjects, R, t);


	//3: Update each src_point
	glm::vec3* src_dev_pos = src_pc->dev_pos;
	for (int i = 0; i < numObjects; ++i) {
		src_dev_pos[i] = glm::transpose(R) * src_dev_pos[i] + t;
	}
}

/**
 * Finds Nearest Neighbors of target pc in src pc
 * @args: src, target -> PointClouds w/ filled dev_pos
 * @returns: 
	* dist -> N array -> ith index = dist(src[i], closest_point in target)
	* indicies -> N array w/ ith index = index of the closest point in target to src[i]
*/
void ScanMatch::findNNCPU(pointcloud* src, pointcloud* target, float* dist, int *indicies, int N) {
	glm::vec3* src_dev_pos = src->dev_pos;
	glm::vec3* target_dev_pos = target->dev_pos;
	for (int src_idx = 0; src_idx < N; ++src_idx) { //Iterate through each source point
		glm::vec3 src_pt = src_dev_pos[src_idx];
		float minDist = INFINITY;
		int idx_minDist = -1;
		for (int tgt_idx = 0; tgt_idx < N; ++tgt_idx) { //Iterate through each tgt point and find closest
			glm::vec3 tgt_pt = target_dev_pos[tgt_idx];
			float d = glm::distance(src_pt, tgt_pt);
			if (d < minDist) {
				minDist = d;
				idx_minDist = tgt_idx;
			}
		}
		//Update dist and indicies

#if DEBUG
		printf("IDX: %d - MINDIST %f\n", src_idx, minDist);
		printf("IDX: %d - indicies %d\n", src_idx, idx_minDist);
#endif // DEBUG

		dist[src_idx] = minDist;
		indicies[src_idx] = idx_minDist;
	}
}

/**
 * Reshuffles pointcloud a as per indicies, puts these in dev_matches
 * NOT ONE TO ONE SO NEED TO MAKE A COPY!
*/
void ScanMatch::reshuffleCPU(pointcloud* a, int* indicies, int N) {
	glm::vec3 *a_dev_matches = a->dev_matches;
	glm::vec3 *a_dev_pos = a->dev_pos;
	for (int i = 0; i < N; ++i) {
		a_dev_matches[i] = a_dev_pos[indicies[i]];

#if DEBUG
		printf("DEV MATCHES\n");
		utilityCore::printVec3(a->dev_matches[i]);
		printf("DEV POS\n");
		utilityCore::printVec3(a_dev_pos[i]);
#endif // DEBUG
	}
}

/**
 * Calculates transform T that maps from src to target
 * Assumes dev_matches is filled for target
*/
void ScanMatch::bestFitTransform(pointcloud* src, pointcloud* target, int N, glm::mat3 &R, glm::vec3 &t){
	glm::vec3* src_norm = new glm::vec3[N];
	glm::vec3* target_norm = new glm::vec3[N];
	glm::vec3 src_centroid(0.f);
	glm::vec3 target_centroid(0.f);
	glm::vec3* src_pos = src->dev_pos;
	glm::vec3* target_matches = target->dev_matches;

	//1:Calculate centroids and norm src and target
	for (int i = 0; i < N; ++i) {
		src_centroid += src_pos[i];
		target_centroid += target_matches[i];
	}
	src_centroid = src_centroid / glm::vec3(N);
	target_centroid = target_centroid / glm::vec3(N);

#if DEBUG
	printf("SRC CENTROID\n");
	utilityCore::printVec3(src_centroid);
	printf("TARGET CENTROID\n");
	utilityCore::printVec3(target_centroid);
#endif // DEBUG

	for (int j = 0; j < N; ++j) {
		src_norm[j] = src_pos[j]  - src_centroid;
		target_norm[j] = target_matches[j] - target_centroid;
#if DEBUG
		printf("SRC NORM IDX %d\n", j);
		utilityCore::printVec3(src_norm[j]);
		printf("TARGET NORM IDX %d\n", j);
		utilityCore::printVec3(target_norm[j]);
#endif // DEBUG
	}

	//1:Multiply src.T (3 x N) by target (N x 3) = H (3 x 3)
	float H[3][3] = { 0 };
	for (int i = 0; i < N; ++i) { //3 x N by N x 3 matmul
		for (int out_row = 0; out_row < 3; out_row++) {
			for (int out_col = 0; out_col < 3; out_col++) {
				H[out_row][out_col] += src_norm[i][out_row] * target_norm[i][out_col];
			}
		}
	}
	
#if DEBUG
	printf("H MATRIX ======================================================\n");
    std::cout << H[0][0] << " " << H[1][0] << " " << H[2][0] << " " << std::endl;
    std::cout << H[0][1] << " " << H[1][1] << " " << H[2][1] << " " << std::endl;
    std::cout << H[0][2] << " " << H[1][2] << " " << H[2][2] << " " << std::endl;
	printf("======================================================\n");
#endif // DEBUG

	//2:calculate SVD of H to get U, S & V
	float U[3][3] = { 0 };
	float S[3][3] = { 0 };
	float V[3][3] = { 0 };
	svd(H[0][0], H[0][1], H[0][2], H[1][0], H[1][1], H[1][2], H[2][0], H[2][1], H[2][2],
		U[0][0], U[0][1], U[0][2], U[1][0], U[1][1], U[1][2], U[2][0], U[2][1], U[2][2],
		S[0][0], S[0][1], S[0][2], S[1][0], S[1][1], S[1][2], S[2][0], S[2][1], S[2][2],
		V[0][0], V[0][1], V[0][2], V[1][0], V[1][1], V[1][2], V[2][0], V[2][1], V[2][2]
		);
	glm::mat3 matU(glm::vec3(U[0][0], U[1][0], U[2][0]), glm::vec3(U[0][1], U[1][1], U[2][1]), glm::vec3(U[0][2], U[1][2], U[2][2]));
	glm::mat3 matV(glm::vec3(V[0][0], V[0][1], V[0][2]), glm::vec3(V[1][0], V[1][1], V[1][2]), glm::vec3(V[2][0], V[2][1], V[2][2]));

#if DEBUG
	printf("U MATRIX\n");
	utilityCore::printMat3(matU);
	printf("V MATRIX\n");
	utilityCore::printMat3(matV);
#endif // DEBUG

	//2:Rotation Matrix and Translation Vector
	R = (matU * matV);
	t = target_centroid - R * (src_centroid);

#if DEBUG
	printf("ROTATION\n");
	utilityCore::printMat3(R);
	printf("TRANSLATION\n");
	utilityCore::printVec3(t);
#endif // DEBUG
}

/******************
* GPU NAIVE SCANMATCHING *
******************/

__global__ void kernUpdatePositions(glm::vec3* src_pos, glm::mat3 R, glm::vec3 t, int N) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (idx < N) {
	  src_pos[idx] = (R) * src_pos[idx] + t;
  }
}

/**
 * Main Algorithm for Running ICP on the GPU
 * Finds homogenous transform between src_pc and target_pc 
*/
void ScanMatch::stepICPGPU_NAIVE() {

	//hipMalloc dist and indicies
	float* dist;
	int* indicies;

	hipMalloc((void**)&dist, numObjects * sizeof(float));
	utilityCore::checkCUDAError("hipMalloc dist failed", __LINE__);

	hipMalloc((void**)&indicies, numObjects * sizeof(int));
	utilityCore::checkCUDAError("hipMalloc indicies failed", __LINE__);
	hipMemset(dist, 0, numObjects * sizeof(float));
	hipMemset(indicies, -1, numObjects * sizeof(int));

	//1: Find Nearest Neigbors and Reshuffle
	ScanMatch::findNNGPU_NAIVE(src_pc, target_pc, dist, indicies, numObjects);
	hipDeviceSynchronize();
	ScanMatch::reshuffleGPU(target_pc, indicies, numObjects);
	hipDeviceSynchronize();
	//2: Find Best Fit Transformation
	glm::mat3 R;
	glm::vec3 t;
	ScanMatch::bestFitTransformGPU(src_pc, target_pc, numObjects, R, t);
	hipDeviceSynchronize();

	//3: Update each src_point via Kernel Call
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
	kernUpdatePositions<<<fullBlocksPerGrid, blockSize>>>(src_pc->dev_pos, R, t, numObjects);

	//hipFree dist and indicies
	hipFree(dist);
	hipFree(indicies);
}

/*
 * Parallely compute NN for each point in the pointcloud
 */
__global__ void kernNNGPU_NAIVE(glm::vec3* src_pos, glm::vec3* target_pos, float* dist, int* indicies, int N) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (idx < N) {
	  float minDist = INFINITY;
	  float idx_minDist = -1;
	  glm::vec3 src_pt = src_pos[idx];
	  for (int tgt_idx = 0; tgt_idx < N; ++tgt_idx) { //Iterate through each tgt & find closest
		  glm::vec3 tgt_pt = target_pos[tgt_idx];
		  float d = glm::distance(src_pt, tgt_pt);
		  //float d = sqrtf(powf((tgt_pt.x - src_pt.x), 2.f) + powf((tgt_pt.y - src_pt.y), 2.f) + powf((tgt_pt.z - src_pt.z), 2.f));
		  if (d < minDist) {
			  minDist = d;
			  idx_minDist = tgt_idx;
		  }
	  }
	  dist[idx] = minDist;
	  indicies[idx] = idx_minDist;
  }
}

/**
 * Finds Nearest Neighbors of target pc in src pc
 * @args: src, target -> PointClouds w/ filled dev_pos IN GPU
 * @returns: 
	* dist -> N array -> ith index = dist(src[i], closest_point in target) (on GPU)
	* indicies -> N array w/ ith index = index of the closest point in target to src[i] (on GPU)
*/
void ScanMatch::findNNGPU_NAIVE(pointcloud* src, pointcloud* target, float* dist, int *indicies, int N) {
	//Launch a kernel (paralellely compute NN for each point)
	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);
	kernNNGPU_NAIVE<<<fullBlocksPerGrid, blockSize>>>(src->dev_pos, target->dev_pos, dist, indicies, N);
}

/*
 * Parallely reshuffle pos by indicies and fill matches
 */
__global__ void kernReshuffleGPU(glm::vec3* pos, glm::vec3* matches, int *indicies, int N) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (idx < N) {
	  //matches[idx] = pos[idx];
	  matches[idx] = pos[indicies[idx]];
  }
}

/**
 * Reshuffles pointcloud a as per indicies, puts these in dev_matches
 * NOT ONE TO ONE SO NEED TO MAKE A COPY!
*/
void ScanMatch::reshuffleGPU(pointcloud* a, int* indicies, int N) {
	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);
	kernReshuffleGPU<<<fullBlocksPerGrid, blockSize>>>(a->dev_pos, a->dev_matches, indicies, N);
}

__global__ void kernComputeNorms(glm::vec3* src_norm, glm::vec3* target_norm, glm::vec3* pos, glm::vec3* matches, glm::vec3 pos_centroid, glm::vec3 matches_centroid, int N) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (idx < N) {
	  src_norm[idx] = pos[idx] - pos_centroid;
	  target_norm[idx] = matches[idx] - matches_centroid;
  }
}

__global__ void kernComputeHarray(glm::mat3* Harray, glm::vec3* src_norm, glm::vec3* target_norm, int N) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (idx < N) {
	  Harray[idx] = glm::mat3(glm::vec3(src_norm[idx]) * target_norm[idx].x,
		  glm::vec3(src_norm[idx]) * target_norm[idx].y,
		  glm::vec3(src_norm[idx]) * target_norm[idx].z);
 }
}

/**
 * Calculates transform T that maps from src to target
 * Assumes dev_matches is filled for target
*/
void ScanMatch::bestFitTransformGPU(pointcloud* src, pointcloud* target, int N, glm::mat3 &R, glm::vec3 &t){

	glm::vec3* src_norm;
	glm::vec3* target_norm;
	glm::mat3* Harray;

	//hipMalloc Norms and Harray
	hipMalloc((void**)&src_norm, N * sizeof(glm::vec3));
	hipMalloc((void**)&target_norm, N * sizeof(glm::vec3));
	hipMalloc((void**)&Harray, N * sizeof(glm::mat3));
	hipMemset(Harray, 0, N * sizeof(glm::mat3));


	//Thrust device pointers for calculating centroids
	thrust::device_ptr<glm::vec3> src_thrustpos(src->dev_pos);
	thrust::device_ptr<glm::vec3> target_thrustmatches(target->dev_matches);
	thrust::device_ptr<glm::mat3> harray_thrust = thrust::device_pointer_cast(Harray);

	//1: Calculate centroids
	glm::vec3 src_centroid(0.f);
	glm::vec3 target_centroid(0.f);
	src_centroid = glm::vec3(thrust::reduce(src_thrustpos, src_thrustpos + N, glm::vec3(0.f), thrust::plus<glm::vec3>()));
	hipDeviceSynchronize();
	target_centroid = glm::vec3(thrust::reduce(target_thrustmatches, target_thrustmatches + N, glm::vec3(0.f), thrust::plus<glm::vec3>()));
	hipDeviceSynchronize();
	src_centroid /= glm::vec3(N);
	target_centroid /= glm::vec3(N);

#if DEBUG
	printf("SRC CENTROID\n");
	utilityCore::printVec3(src_centroid);
	printf("TARGET CENTROID\n");
	utilityCore::printVec3(target_centroid);
#endif // DEBUG

	//2: Compute Norm via Kernel Call
	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);
	kernComputeNorms<<<fullBlocksPerGrid, blockSize>>>(src_norm, target_norm, src->dev_pos, target->dev_matches, src_centroid, target_centroid, N);
	hipDeviceSynchronize();
	utilityCore::checkCUDAError("Compute Norms Failed", __LINE__);

	//3:Multiply src.T (3 x N) by target (N x 3) = H (3 x 3) via a kernel call
	kernComputeHarray<<<fullBlocksPerGrid, blockSize>>>(Harray, src_norm, target_norm, N);
	hipDeviceSynchronize();
	utilityCore::checkCUDAError("Compute HARRAY Failed", __LINE__);

	/*
	glm::mat3 H = thrust::reduce(harray_thrust, harray_thrust + N, glm::mat3(0.f), thrust::plus<glm::mat3>());
	hipDeviceSynchronize();
	*/

	glm::mat3* Hcpu = new glm::mat3[N];
	hipMemcpy(Hcpu, Harray, N * sizeof(glm::mat3), hipMemcpyDeviceToHost);
	utilityCore::checkCUDAError("REDUCE HARRAY Failed", __LINE__);
	hipDeviceSynchronize();
	glm::mat3 H(0.f);
	for (int i = 0; i < N; ++i) {
		H += Hcpu[i];
	}
	//4:Calculate SVD of H to get U, S & V
	float U[3][3] = { 0 };
	float S[3][3] = { 0 };
	float V[3][3] = { 0 };
	svd(H[0][0], H[0][1], H[0][2], H[1][0], H[1][1], H[1][2], H[2][0], H[2][1], H[2][2],
		U[0][0], U[0][1], U[0][2], U[1][0], U[1][1], U[1][2], U[2][0], U[2][1], U[2][2],
		S[0][0], S[0][1], S[0][2], S[1][0], S[1][1], S[1][2], S[2][0], S[2][1], S[2][2],
		V[0][0], V[0][1], V[0][2], V[1][0], V[1][1], V[1][2], V[2][0], V[2][1], V[2][2]
		);
	glm::mat3 matU(glm::vec3(U[0][0], U[1][0], U[2][0]), glm::vec3(U[0][1], U[1][1], U[2][1]), glm::vec3(U[0][2], U[1][2], U[2][2]));
	glm::mat3 matV(glm::vec3(V[0][0], V[0][1], V[0][2]), glm::vec3(V[1][0], V[1][1], V[1][2]), glm::vec3(V[2][0], V[2][1], V[2][2]));

	//5:Rotation Matrix and Translation Vector
	R = (matU * matV);
	t = target_centroid - (R) * (src_centroid);

	//hipMalloc Norms and Harray
	hipFree(src_norm);
	hipFree(target_norm); 
	hipFree(Harray);
}
