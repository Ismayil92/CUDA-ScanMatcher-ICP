#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "scanmatch.h"

#define MAX_ICP_ITERS 1000
#define EPSILON 0.001

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) utilityCore::checkCUDAError(msg, __LINE__)

/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

/*! Size of the starting area in simulation space. 
 * FOR SINE TEST: 2.f
 * FOR ELEPHANT OBJ: 
 * FOR WAYMO DATASET: 
*/

#define scene_scale 2.f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

glm::vec3 *dev_pos;
glm::vec3 *dev_rgb;

pointcloud* target_pc;
pointcloud* src_pc;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* Initialize memory, update some globals
*/
void ScanMatch::initSimulation(int N) {
  numObjects = N;

  //Setup and initialize source and target pointcloud
  src_pc = new pointcloud(false, numObjects);
  src_pc->initCPU();

  target_pc = new pointcloud(true, numObjects);
  target_pc->initCPU();
}

/******************
* copyPointCloudToVBO *
******************/

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void ScanMatch::copyPointCloudToVBO(float *vbodptr_positions, float *vbodptr_rgb) {

  //IF CPU
  src_pc->pointCloudToVBOCPU(vbodptr_positions, vbodptr_rgb, scene_scale);
  target_pc->pointCloudToVBOCPU(vbodptr_positions + 4*numObjects, vbodptr_rgb + 4*numObjects, scene_scale);
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
  // Rule 2: boids try to stay a distance d away from each other
  // Rule 3: boids try to match the speed of surrounding boids
  return glm::vec3(0.0f, 0.0f, 0.0f);
}

__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
  // Clamp the speed
  // Record the new velocity into vel2. Question: why NOT vel1?
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
}

void ScanMatch::endSimulation() {
	src_pc->~pointcloud();
	target_pc->~pointcloud();
}

/******************
* CPU SCANMATCHING *
******************/

/**
 * Main Algorithm for Running ICP on the CPU
 * Finds homogenous transform between src_pc and target_pc 
*/
void ScanMatch::ICPCPU() {
	for (int i = 0; i < MAX_ICP_ITERS; ++i) {
		//1: Find Nearest Neigbors and Reshuffle
		float* dist = new float[numObjects];
		int* indicies = new int[numObjects];
		//ScanMatch::findNNCPU(src_pc, target_pc, dist, indicies, numObjects);
		//reshuffleCPU(target_pc, indicies, numObjects);
	}
}

